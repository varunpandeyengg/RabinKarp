#include "hip/hip_runtime.h"
/*
 * rabin_karp_v3.cpp
 * 
 * Created by Varun Pandey on 05-19-2015.
 * 
 * Copyright (c) 2015 Varun Pandey
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 
 * Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 * 
 * redistributions in binary form must reproduce the above copyright
 * notice, this list of conditions and the following disclaimer in the
 * documentation and/or other materials provided with the distribution.
 * 
 * Neither the name of the project's author nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
 * TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
 * LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "../include/common_headers.h"
#include "../include/constant.h"

__device__ bool d_found;

__device__
int remove_byte(int64_t& current_digest, BYTE from_digest, size_t msb_pos_window);
__device__
int insert_byte(int64_t& current_digest, BYTE from_digest);
/* Cleanup rutine before successfull termination of this program
 * */
void OnExit (void) 
{
	std::cout << std::endl << "Terminating Now..." << std::endl;
}

/*
 * Cuda Kernel that will be used run independently on 2*(size of pattern)
 * to find the patern.
 * Returns true on success.
 * */
__global__ 
void find_pattern (BYTE* data, int64_t pattern_fingerprint, size_t window_size, size_t msb_multiplier)
{
	int msb_pos_window = 0;
	int64_t text_digest = 0;
	int start_pos = threadIdx.x;
	BYTE *window = new BYTE [2*window_size];
	if (!window)
		return;

	if (start_pos % (window_size*2))
	{
		/*
 		* start with adding window_size bytes to the digest
 		* and try to matching the fingerprint for window_size+1 shifts
 		* */
		int i = 0;
		for (; i < window_size; i++) {
			insert_byte (text_digest, *(data+start_pos+i));
			window[i] = *(data+start_pos+i);

		}
			
		for (++i; i < (2*window_size); i++)
		{
			if (text_digest == pattern_fingerprint) {
				d_found = true; break;
			}
			
			remove_byte (text_digest, window[msb_pos_window++], msb_multiplier);
			window[i] = *(data+start_pos+i);
			insert_byte (text_digest, *(data+start_pos+i));
		}
	}
}
/* Assuing that the bytes is in the msb position, removes 
 * the byte.
 * returns 0 on success
 * */
__device__
int remove_byte(int64_t& current_digest, BYTE from_digest, size_t msb_multiplier)
{
	int ret_val = 0;

	current_digest = current_digest - (msb_multiplier * from_digest); //shift the byte
	/*After much head banging, I am adding this code. Underflowing hash, damn it!*/
	while (current_digest < 0)
		current_digest += PRIME;
	
	return ret_val;
}

/* Insert single byte into the digest 
 * Note: - The first byte will goto msb position. So data[0]
 * is at the highest order of the polynomial. Subsequently,
 * this is the first byte to be kicked out of our window.
 * Window size = pattern size (for now). For large patterns
 * proportional to text, window size can be controlled.
 * returns 0 on success
 *
 * */
__device__
int insert_byte(int64_t& current_digest, BYTE to_digest)
{
	int ret_val = 0;

	current_digest *= RADIX; //shift the byte
	current_digest += to_digest;
	current_digest %= PRIME;
	
	return ret_val;
}

int insert_byte_host(int64_t& current_digest, BYTE to_digest, size_t msb_pos_window)
{
        int ret_val = 0;

        current_digest *= RADIX; //shift the byte
        current_digest += to_digest;
        current_digest %= PRIME;

        return ret_val;
}

/* Process $size bytes and add it into the into the digest
 * returns 0 on success
 * */
int insert_bytes(int64_t& current_digest, BYTE* data, size_t size)
{
	int ret_val = 0;
	for (int i = (size_t)0; i < size; i++){
		insert_byte_host (current_digest, data[i], msb_multiplier);		
	}

	return ret_val;
}

int main (int argc, char ** argv)
{
	typeof(d_found) found = false;
	size_t i = 0;
	int64_t pattern_hash = 0;
		
	BYTE* read_buf = NULL;
	BYTE* window = NULL; 

	std::ifstream file_stream;
	std::streampos stream_pos;
	
	if (argc != 3)
		USAGE_EXIT(argv[0]);
	
	atexit (OnExit);	

	try {
		window_size = strlen(argv[2]); //Pattern size = window size
		for (i = 1; i < window_size; i++)
			msb_multiplier = (msb_multiplier*RADIX)%PRIME;

		// Check if file exist		
		file_stream.open (argv[1], std::ios::in | std::ios::binary);
		if (file_stream.bad() || file_stream.fail())
			THROW("Please check the file path and try again");
	
		//Get File size
       		size_t fsize = file_stream.tellg();
       		file_stream.seekg (0, std::ios::end);
	    	fsize = file_stream.tellg() - (std::streampos)fsize;
		if (fsize % window_size) 
			fsize = fsize + (window_size - (fsize % window_size)); //create extra room for padding if needed
		
		//reset stream
		file_stream.clear();                 // clear fail and eof bits
		file_stream.seekg(0, std::ios::beg); // back to the start!

		read_buf = new BYTE [fsize];
                if (!read_buf)
                        THROW("Bad allocation");
		memset (read_buf, '@', fsize); // Pad the data with @
	
		//Fill the window and get the 
	        file_stream.read(read_buf, fsize);
		insert_bytes (pattern_hash, argv[2], window_size);
		
#ifdef DEBUG	/*For Dedugging*/
		std::cout << std::endl << "Pattern hash: " << pattern_hash << std::endl;
#endif
			
		BYTE* d_text;
		hipMalloc ((void**)&d_text, fsize); 
		hipMemcpy(d_text, read_buf, fsize, hipMemcpyHostToDevice );
		
		find_pattern<<<1, 1024>>>(d_text, pattern_hash, window_size, msb_multiplier);
		
		hipMemcpyFromSymbol(&found, HIP_SYMBOL(d_found), sizeof(found), hipMemcpyDeviceToHost);
		hipFree(d_text);
	} catch (const std::exception& e) {
		REPORT(e);
	}

	/*Display Result*/
	PATTERN_REPORT(found, file_stream.tellg());
	
	/*Cleanup*/
	if (read_buf)	
		delete [] read_buf, read_buf = NULL;
	if (window)
		delete [] window, window = NULL;

	file_stream.close();
	
	return 0;
}
